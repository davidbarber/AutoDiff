
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void vcopyfrom_update(const int n, const int shift, const double *a, double *b)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
      {
	b[i] += a[i+shift];
      }
  }
}