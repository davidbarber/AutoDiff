
#include <hip/hip_runtime.h>
// filename: ax.cu
// a simple CUDA kernel to add two vectors

extern "C"   // ensure function name to be exactly "ax"
{
    __global__ void ax(const int lengthC, const double a, const double *b, double *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthC)
	{
        c[i] = a*b[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
	}
    }
}