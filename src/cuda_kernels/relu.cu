
#include <hip/hip_runtime.h>
extern "C"  
{
  __global__ void relu(const int n, const double *a, double *b)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<n)
      {
	if (a[i]>0.0)
	  {b[i] = a[i];}	
	else	 
	  {b[i] = 0.0;}
      }	
  }
}